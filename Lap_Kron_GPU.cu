#include <stdio.h>
#include <assert.h>
#include <mkl.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hip/hip_complex.h>

#include "Lap_Kron_GPU.h"
#include "tools.h"

#ifdef __cplusplus
extern "C" {
#endif

void CUDA_Lap_Kron(int Nx, int Ny, int Nz, double *d_Vx, double *d_Vy, double *d_Vz, 
                 double *d_vec, double *d_diag, double *d_out)
{
	hipblasHandle_t handle;
	hipblasStatus_t cubSt;
	hipError_t cuE;

	cubSt = hipblasCreate(&handle); assert(HIPBLAS_STATUS_SUCCESS == cubSt);

	int NxNy = Nx * Ny;
    int Nd = Nx * Ny * Nz;
	double *d_vecTVy, *d_VxtvecTVy, *d_P, *d_PTVyt, *d_VxPTVyt;
	cuE = hipMalloc((void **) &d_vecTVy, sizeof(double) * NxNy); assert(hipSuccess == cuE);
	d_VxtvecTVy = d_out;
	cuE = hipMalloc((void **) &d_P, sizeof(double) * Nd); assert(hipSuccess == cuE);
	d_PTVyt = d_vecTVy;
	d_VxPTVyt = d_P;

	double alpha = 1.0, beta = 0;
    for (int k = 0; k < Nz; k++) {
		cubSt = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Nx, Ny, Ny, 
					&alpha, d_vec + k*NxNy, Nx, d_Vy, Ny, &beta, d_vecTVy, Nx); 
		assert(HIPBLAS_STATUS_SUCCESS == cubSt);
		
        cubSt = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, Nx, Ny, Nx, 
                    &alpha, d_Vx, Nx, d_vecTVy, Nx, &beta, d_VxtvecTVy + k*NxNy, Nx);
		assert(HIPBLAS_STATUS_SUCCESS == cubSt);
    }

	cubSt = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, NxNy, Nz, Nz, 
                    &alpha, d_VxtvecTVy, NxNy, d_Vz, Nz, &beta, d_P, NxNy);
	assert(HIPBLAS_STATUS_SUCCESS == cubSt);

	int numThreadsPerBlock = 256;
	int numBlocks = (Nd + numThreadsPerBlock - 1) / numThreadsPerBlock;

	Hammond_RR<<<numBlocks, numThreadsPerBlock>>>(d_P, d_diag, d_P, Nd);

	for (int k = 0; k < Nz; k++) {
		cubSt = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, Nx, Ny, Ny, 
                    &alpha, d_P + k*NxNy, Nx, d_Vy, Ny, &beta, d_PTVyt, Nx);
		assert(HIPBLAS_STATUS_SUCCESS == cubSt);

		cubSt = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Nx, Ny, Nx, 
                    &alpha, d_Vx, Nx, d_PTVyt, Nx, &beta, d_VxPTVyt + k*NxNy, Nx);
		assert(HIPBLAS_STATUS_SUCCESS == cubSt);
    }
    
	cubSt = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, NxNy, Nz, Nz, 
                    &alpha, d_VxPTVyt, NxNy, d_Vz, Nz, &beta, d_out, NxNy);
	assert(HIPBLAS_STATUS_SUCCESS == cubSt);
	
	cuE = hipFree(d_vecTVy); assert(hipSuccess == cuE);
	cuE = hipFree(d_P); assert(hipSuccess == cuE);
	cubSt = hipblasDestroy(handle); assert(HIPBLAS_STATUS_SUCCESS == cubSt);
}


void CUDA_Lap_Kron_complex(int Nx, int Ny, int Nz, hipDoubleComplex *d_Vx, hipDoubleComplex *d_Vy, hipDoubleComplex *d_Vz, 
                 hipDoubleComplex *d_VyH, hipDoubleComplex *d_VzH, hipDoubleComplex *d_vec, double *d_diag, hipDoubleComplex *d_out)
{
	hipblasHandle_t handle;
	hipblasStatus_t cubSt;
	hipError_t cuE;

	cubSt = hipblasCreate(&handle); assert(HIPBLAS_STATUS_SUCCESS == cubSt);

	int NxNy = Nx * Ny;
    int Nd = Nx * Ny * Nz;
	
	hipDoubleComplex *d_vecTVy, *d_VxtvecTVy, *d_P, *d_PTVyt, *d_VxPTVyt;
	cuE = hipMalloc((void **) &d_vecTVy, sizeof(hipDoubleComplex) * NxNy); assert(hipSuccess == cuE);
	d_VxtvecTVy = d_out;    
	cuE = hipMalloc((void **) &d_P, sizeof(hipDoubleComplex) * Nd); assert(hipSuccess == cuE);    
	d_PTVyt = d_vecTVy;    
	d_VxPTVyt = d_P;    
    hipDoubleComplex aplha = make_hipDoubleComplex(1.0, 0.0), beta = make_hipDoubleComplex(0.0, 0.0);

    // P = Lambda .* (VzH x VyH x VxH) * vec
    for (int k = 0; k < Nz; k++) {    
		cubSt = hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Nx, Ny, Ny, 
                    &aplha, d_vec + k*NxNy, Nx, d_VyH, Ny, &beta, d_vecTVy, Nx);
		assert(HIPBLAS_STATUS_SUCCESS == cubSt);
    
	    cubSt = hipblasZgemm(handle, HIPBLAS_OP_C, HIPBLAS_OP_N, Nx, Ny, Nx, 
                    &aplha, d_Vx, Nx, d_vecTVy, Nx, &beta, d_VxtvecTVy + k*NxNy, Nx);
		assert(HIPBLAS_STATUS_SUCCESS == cubSt);
    }

	cubSt = hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, NxNy, Nz, Nz, 
                    &aplha, d_VxtvecTVy, NxNy, d_VzH, Nz, &beta, d_P, NxNy);
	assert(HIPBLAS_STATUS_SUCCESS == cubSt);

	int numThreadsPerBlock = 256;
	int numBlocks = (Nd + numThreadsPerBlock - 1) / numThreadsPerBlock;

	Hammond_CR<<<numBlocks, numThreadsPerBlock>>>(d_P, d_diag, d_P, Nd);

    // out = (Vz x Vy x Vx) * P
    for (int k = 0; k < Nz; k++) {
	    cubSt = hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, Nx, Ny, Ny, 
                    &aplha, d_P + k*NxNy, Nx, d_Vy, Ny, &beta, d_PTVyt, Nx);
		assert(HIPBLAS_STATUS_SUCCESS == cubSt);
    
	    cubSt = hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Nx, Ny, Nx, 
                    &aplha, d_Vx, Nx, d_PTVyt, Nx, &beta, d_VxPTVyt + k*NxNy, Nx);
		assert(HIPBLAS_STATUS_SUCCESS == cubSt);
    }

    cubSt = hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, NxNy, Nz, Nz, 
                    &aplha, d_VxPTVyt, NxNy, d_Vz, Nz, &beta, d_out, NxNy);
	assert(HIPBLAS_STATUS_SUCCESS == cubSt);

	cuE = hipFree(d_vecTVy); assert(hipSuccess == cuE);
	cuE = hipFree(d_P); assert(hipSuccess == cuE);
	cubSt = hipblasDestroy(handle); assert(HIPBLAS_STATUS_SUCCESS == cubSt);
}

#ifdef __cplusplus
}
#endif