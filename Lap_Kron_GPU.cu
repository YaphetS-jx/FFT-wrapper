#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <assert.h>
#include <mkl.h>

#include "Lap_Kron_GPU.h"

#ifdef __cplusplus
extern "C" {
#endif

__global__ void elementWiseMultiply(double* a, double* b, double* c, int n) 
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] * b[i];
    }
}


void Lap_Kron_CUDA(int Nx, int Ny, int Nz, double *d_Vx, double *d_Vy, double *d_Vz, 
                 double *d_vec, double *d_diag, double *d_out)
{
	hipblasHandle_t handle;
	hipblasStatus_t cubSt;
	hipError_t cuE;

	cubSt = hipblasCreate(&handle); assert(HIPBLAS_STATUS_SUCCESS == cubSt);

	int NxNy = Nx * Ny;
    int Nd = Nx * Ny * Nz;
	double *d_vecTVy, *d_VxtvecTVy, *d_P, *d_PTVyt, *d_VxPTVyt;
	cuE = hipMalloc((void **) &d_vecTVy, sizeof(double) * NxNy); assert(hipSuccess == cuE);
	d_VxtvecTVy = d_out;
	cuE = hipMalloc((void **) &d_P, sizeof(double) * Nd); assert(hipSuccess == cuE);
	d_PTVyt = d_vecTVy;
	d_VxPTVyt = d_P;

	double alpha = 1.0, beta = 0;
    for (int k = 0; k < Nz; k++) {
		cubSt = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Nx, Ny, Ny, 
					&alpha, d_vec + k*NxNy, Nx, d_Vy, Ny, &beta, d_vecTVy, Nx); 
		assert(HIPBLAS_STATUS_SUCCESS == cubSt);
		
        cubSt = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, Nx, Ny, Nx, 
                    &alpha, d_Vx, Nx, d_vecTVy, Nx, &beta, d_VxtvecTVy + k*NxNy, Nx);
		assert(HIPBLAS_STATUS_SUCCESS == cubSt);
    }

	cubSt = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, NxNy, Nz, Nz, 
                    &alpha, d_VxtvecTVy, NxNy, d_Vz, Nz, &beta, d_P, NxNy);
	assert(HIPBLAS_STATUS_SUCCESS == cubSt);

	int numThreadsPerBlock = 256;
	int numBlocks = (Nd + numThreadsPerBlock - 1) / numThreadsPerBlock;

	elementWiseMultiply<<<numBlocks, numThreadsPerBlock>>>(d_P, d_diag, d_P, Nd);

	for (int k = 0; k < Nz; k++) {
		cubSt = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, Nx, Ny, Ny, 
                    &alpha, d_P + k*NxNy, Nx, d_Vy, Ny, &beta, d_PTVyt, Nx);
		assert(HIPBLAS_STATUS_SUCCESS == cubSt);

		cubSt = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Nx, Ny, Nx, 
                    &alpha, d_Vx, Nx, d_PTVyt, Nx, &beta, d_VxPTVyt + k*NxNy, Nx);
		assert(HIPBLAS_STATUS_SUCCESS == cubSt);
    }
    
	cubSt = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, NxNy, Nz, Nz, 
                    &alpha, d_VxPTVyt, NxNy, d_Vz, Nz, &beta, d_out, NxNy);
	assert(HIPBLAS_STATUS_SUCCESS == cubSt);
	
	cuE = hipFree(d_vecTVy); assert(hipSuccess == cuE);
	cuE = hipFree(d_P); assert(hipSuccess == cuE);

	cubSt = hipblasDestroy(handle); assert(HIPBLAS_STATUS_SUCCESS == cubSt);
}



#ifdef __cplusplus
}
#endif