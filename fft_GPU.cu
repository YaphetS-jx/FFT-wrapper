#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "hipblas.h"
#include <hipfft/hipfft.h>

#include "fft_GPU.h"
#include "tools.h"


#ifdef __cplusplus
extern "C" {
#endif



void CUDA_fft_solve(int Nx, int Ny, int Nz, double *d_rhs, double *d_pois_FFT_const, double *d_sol)
{
    int Nd_half = (Nx/2+1)*Ny*Nz;
    int dim_sizes[3] = {Nz, Ny, Nx};

    hipError_t cuE;

    hipDoubleComplex *d_rhs_bar;
    cuE = hipMalloc((void **) &d_rhs_bar, sizeof(hipDoubleComplex) * Nd_half); assert(hipSuccess == cuE);

    CUDA_MDFFT_real(d_rhs, dim_sizes, d_rhs_bar);  

    int numThreadsPerBlock = 256;
	int numBlocks = (Nd_half + numThreadsPerBlock - 1) / numThreadsPerBlock;

	Hammond_CR<<<numBlocks, numThreadsPerBlock>>>(d_rhs_bar, d_pois_FFT_const, d_rhs_bar, Nd_half);

    CUDA_MDiFFT_real(d_rhs_bar, dim_sizes, d_sol);

    cuE = hipFree(d_rhs_bar); assert(hipSuccess == cuE);
}


/**
 * @brief   CUDA multi-dimension FFT interface, real to complex, following conjugate even distribution. 
 */
void CUDA_MDFFT_real(hipfftDoubleReal *d_r2c_3dinput, int *dim_sizes, hipfftDoubleComplex *d_r2c_3doutput)
{
    hipfftHandle plan_r2c;
    hipfftCreate(&plan_r2c);
    hipfftPlan3d(&plan_r2c, dim_sizes[0], dim_sizes[1], dim_sizes[2], HIPFFT_D2Z);
    hipfftExecD2Z(plan_r2c, d_r2c_3dinput, d_r2c_3doutput);
    hipfftDestroy(plan_r2c);
}

/**
 * @brief   CUDA multi-dimension FFT interface, complex to real, following conjugate even distribution. 
 *          Warning: d_c2r_3dinput will be changed!
 */
void CUDA_MDiFFT_real(hipfftDoubleComplex *d_c2r_3dinput, int *dim_sizes, hipfftDoubleReal *d_c2r_3doutput)
{
    hipfftHandle plan_c2r;
    hipfftCreate(&plan_c2r);
    hipfftPlan3d(&plan_c2r, dim_sizes[0], dim_sizes[1], dim_sizes[2], HIPFFT_Z2D);
    hipfftExecZ2D(plan_c2r, d_c2r_3dinput, d_c2r_3doutput);
    hipfftDestroy(plan_c2r);

    int N = dim_sizes[2]*dim_sizes[1]*dim_sizes[0];
    int numThreadsPerBlock = 256;
    int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;
    double scale = 1.0/N;
    scale_vector<<<numBlocks,numThreadsPerBlock>>>(d_c2r_3doutput, scale, N);
}


/**
 * @brief   CUDA multi-dimension FFT interface, complex to complex
 */
void CUDA_MDFFT(hipfftDoubleComplex *d_c2c_3dinput, int *dim_sizes, hipfftDoubleComplex *d_c2c_3doutput)
{
    hipfftHandle plan_c2c;
    hipfftCreate(&plan_c2c);
    hipfftPlan3d(&plan_c2c, dim_sizes[0], dim_sizes[1], dim_sizes[2], HIPFFT_Z2Z);
    hipfftExecZ2Z(plan_c2c, d_c2c_3dinput, d_c2c_3doutput, HIPFFT_FORWARD);
    hipfftDestroy(plan_c2c);
}

/**
 * @brief   CUDA multi-dimension FFT interface, complex to complex
 */
void CUDA_MDiFFT(hipfftDoubleComplex *d_c2c_3dinput, int *dim_sizes, hipfftDoubleComplex *d_c2c_3doutput)
{
    hipfftHandle plan_c2c;
    hipfftCreate(&plan_c2c);
    hipfftPlan3d(&plan_c2c, dim_sizes[0], dim_sizes[1], dim_sizes[2], HIPFFT_Z2Z);
    hipfftExecZ2Z(plan_c2c, d_c2c_3dinput, d_c2c_3doutput, HIPFFT_BACKWARD);
    hipfftDestroy(plan_c2c);

    int N = dim_sizes[2]*dim_sizes[1]*dim_sizes[0];
    int numThreadsPerBlock = 256;
    int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;
    double scale = 1.0/N;
    scale_vector_complex<<<numBlocks,numThreadsPerBlock>>>(d_c2c_3doutput, scale, N);
}

// batched FFT
/**
 * @brief   CUDA multi-dimension batch FFT interface, real to complex, following conjugate even distribution. 
 */
void CUDA_MDFFT_batch_real(hipfftDoubleReal *d_r2c_3dinput, int *dim_sizes, int ncol, hipfftDoubleComplex *d_r2c_3doutput)
{
    int N = dim_sizes[2]*dim_sizes[1]*dim_sizes[0];

    hipfftHandle plan_r2c;
    hipfftCreate(&plan_r2c);
    hipfftPlanMany(&plan_r2c, 3, dim_sizes, NULL, 1, N, NULL, 1, N, HIPFFT_D2Z, ncol);
    // hipfftPlanMany(&plan_r2c, 3, dim_sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z, ncol);
    hipfftExecD2Z(plan_r2c, d_r2c_3dinput, d_r2c_3doutput);
    hipfftDestroy(plan_r2c);
}


/**
 * @brief   CUDA multi-dimension batch FFT interface, complex to real, following conjugate even distribution. 
 *          Warning: d_c2r_3dinput will be changed!
 */
void CUDA_MDiFFT_batch_real(hipfftDoubleComplex *d_c2r_3dinput, int *dim_sizes, int ncol, hipfftDoubleReal *d_c2r_3doutput)
{
    int N = dim_sizes[2]*dim_sizes[1]*dim_sizes[0];

    hipfftResult_t res_t;

    hipfftHandle plan_c2r;
    hipfftCreate(&plan_c2r);
    // hipfftPlanMany(&plan_c2r, 3, dim_sizes, NULL, 1, N, NULL, 1, N, HIPFFT_Z2D, ncol);
    res_t = hipfftPlanMany(&plan_c2r, 3, dim_sizes, NULL, 1, N, NULL, 1, N, HIPFFT_Z2D, ncol);
    assert(res_t == HIPFFT_SUCCESS);

    res_t = hipfftExecZ2D(plan_c2r, d_c2r_3dinput, d_c2r_3doutput);
    assert(res_t == HIPFFT_SUCCESS);
    
    hipfftDestroy(plan_c2r);

    int numThreadsPerBlock = 256;
    int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;
    double scale = 1.0/N;
    scale_vector<<<numBlocks,numThreadsPerBlock>>>(d_c2r_3doutput, scale, N*ncol);
}


#ifdef __cplusplus
}
#endif