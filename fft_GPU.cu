#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "hipblas.h"
#include <hipfft/hipfft.h>

#include "fft_GPU.h"


#ifdef __cplusplus
extern "C" {
#endif


__global__ void scale_vector(double* a, double scale, int n) 
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        a[i] *= scale;
    }
}

__global__ void GPU_print_kernel(double *d_vec, int n) 
{
    for (int i = 0; i < n; i++) {
        printf("%20.16f\n", d_vec[i]);
    }
    printf("\n");
}

__global__ void GPU_print_complex_kernel(hipDoubleComplex *d_vec, int n) 
{
    for (int i = 0; i < n; i++) {
        printf("%20.16f + %20.16f\n", hipCreal(d_vec[i]), hipCimag(d_vec[i]));
    }
    printf("\n");
}

void GPU_print(double *d_vec, int n)
{
    GPU_print_kernel<<<1,1>>>(d_vec, n);
}

void GPU_print_complex(hipDoubleComplex *d_vec, int n)
{
    GPU_print_complex_kernel<<<1,1>>>(d_vec, n);
}

void CUDA_MDFFT_real(hipfftDoubleReal *d_r2c_3dinput, int *dim_sizes, hipfftDoubleComplex *d_r2c_3doutput)
{
    hipfftHandle plan_r2c;
    hipfftCreate(&plan_r2c);
    hipfftPlan3d(&plan_r2c, dim_sizes[0], dim_sizes[1], dim_sizes[2], HIPFFT_D2Z);
    hipfftExecD2Z(plan_r2c, d_r2c_3dinput, d_r2c_3doutput);
    hipfftDestroy(plan_r2c);
}

void CUDA_MDiFFT_real(hipfftDoubleComplex *d_c2r_3dinput, int *dim_sizes, hipfftDoubleReal *d_c2r_3doutput)
{
    // out-place C2R will change input 
    int N_half = (dim_sizes[2]/2+1) * dim_sizes[1] * dim_sizes[0];
    hipfftDoubleComplex *d_c2r_3dinput_copy;
    hipMalloc((void **) &d_c2r_3dinput_copy, sizeof(hipfftDoubleComplex) * N_half);
    hipMemcpy(d_c2r_3dinput_copy, d_c2r_3dinput, sizeof(hipfftDoubleComplex)*N_half, hipMemcpyDeviceToDevice);

    hipfftHandle plan_c2r;
    hipfftCreate(&plan_c2r);
    hipfftPlan3d(&plan_c2r, dim_sizes[0], dim_sizes[1], dim_sizes[2], HIPFFT_Z2D);
    hipfftExecZ2D(plan_c2r, d_c2r_3dinput_copy, d_c2r_3doutput);
    hipfftDestroy(plan_c2r);

    int N = dim_sizes[2]*dim_sizes[1]*dim_sizes[0];
    int numThreadsPerBlock = 256;
    int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;
    double scale = 1.0/N;
    scale_vector<<<numBlocks,numThreadsPerBlock>>>(d_c2r_3doutput, scale, N);

    hipFree(d_c2r_3dinput_copy);
}



#ifdef __cplusplus
}
#endif